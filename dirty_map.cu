//compiling line: nvcc --shared -o dms.so --compiler-options -fPIC dirty_map.cu
#include <hip/hip_runtime.h>

#include <math.h>       /* sin, cos, fmod, fabs, asin, atan2 */
#include <stdio.h>
#include <iostream>

#define PI 3.14159265358979323846
#define omega 2*PI/86400 //earth angular velocity in rads/second

struct floatArray {
    float * p;
    unsigned int l;
};

struct chordParams
{
    floatArray thetas;
    float initial_phi_offset; //amount that the calculation starts away from each source
    unsigned int m1; //north south number of dishes
    unsigned int m2; //east west
    float L1; // north osuth dish separation
    float L2; //east west
    float CHORD_zenith_dec;
    float D; //dish diameter
    float delta_tau;
    unsigned int time_samples;
};

__device__ inline void ang2vec (const float theta, const float phi, float outvec [3])
{
    outvec[0] = sin(theta)*cos(phi);
    outvec[1] = sin(theta)*sin(phi);
    outvec[2] = cos(theta);
}

__device__ inline void cross (const float v1 [3], const float v2 [3], float outvec [3])
{
    outvec[0] = v1[1]*v2[2]-v1[2]*v2[1];
    outvec[1] = v1[2]*v2[0]-v1[0]*v2[2];
    outvec[2] = v1[0]*v2[1]-v1[1]*v2[0]; 
}

__device__ inline void rotate (const float v [3], float outvec [3], const float alpha)
{
    outvec[0] = cos(alpha)*v[0] - sin(alpha)*v[1];
    outvec[1] = sin(alpha)*v[0] + cos(alpha)*v[1];
    outvec[2] = v[2];
}

__device__ inline float dot (const float v1 [3], const float v2 [3])
{
    return v1[0]*v2[0]+v1[1]*v2[1]+v1[2]*v2[2];
}

__device__ inline float crossmag(const float v1 [3], const float v2 [3])
{
    float cv [3];
    cross(v1,v2,cv);
    return sqrt(dot(cv,cv));
}

__device__ float B_sq (const float alpha, const float wavelength, const float D)
{
    float alphaprime = PI*D*sin(alpha)/wavelength;
    if (alphaprime <= 1E-8 && alphaprime >= -1E-8)
        return (j0f(alphaprime)-jnf(2,alphaprime))*(j0f(alphaprime)-jnf(2,alphaprime)); //l'Hopital's
    else
        return (2*j1f(alphaprime)/alphaprime) * (2*j1f(alphaprime)/alphaprime);
}

__device__ inline float Bsq_from_vecs (const float v1 [3], const float v2 [3], const float wavelength, const float D)
{
    float dp = dot(v1,v2);
    if (dp <= 0) return 0; //horizon condition
    else
    {
        //we want to deal with the arccos instiblity by using the cross product formula instead
        float delta_ang;
        if (dp < 0.99) delta_ang = std::acos(dp);    
        else 
        {
            delta_ang = std::asin(crossmag(v1,v2));
            //delta_ang = (dp > 0) ? delta_ang : PI-delta_ang; //I don't need this line with the horizon condition
        }
        return B_sq(delta_ang, wavelength, D);
    }
}

__device__ inline float subtractdot (const float v1_a [3], const float v1_b [3], const float v2 [3])
{
    return (v1_a[0]-v1_b[0])*v2[0]+(v1_a[1]-v1_b[1])*v2[1]+(v1_a[2]-v1_b[2])*v2[2];
}

__device__ float sin_sq_ratio (const unsigned int m, const float x_prime)
{
    float x = fmodf(x_prime,PI); // -pi < x < pi
    x = fabs(x); // 0 < x < pi
    x = (x > PI/2) ? PI-x : x; //0 < x < pi/2
    
    if (fabs(x) < 1E-9) return m*m*cos(m*x)*cos(m*x)/(cos(x)*cos(x));
    else return sin(m*x)*sin(m*x)/(sin(x)*sin(x));
}

__global__ void dirtymap_kernel (const floatArray u, const floatArray wavelengths, const floatArray source_positions, const floatArray source_spectra, float brightness_threshold, const chordParams cp, float * dm)
{
    //printf("%d ", blockIdx.x*32 + threadIdx.x);
    int deviceID;
    hipGetDevice(&deviceID);
    //if (blockIdx.x*32 + threadIdx.x == 0) printf("hello, I'm running from device %d\n", deviceID);
    if ((blockIdx.x*32 + threadIdx.x)*3 < u.l)
    {
        //calculating the relevant CHORD vectors for each dither direction
        float * chord_pointing = new float [3*cp.thetas.l];
        float * dir1_proj_vec = new float [3*cp.thetas.l]; //north/south chord direction
        float * dir2_proj_vec = new float [3*cp.thetas.l]; //east/west chord direction
        for (unsigned int k = 0; k < cp.thetas.l; k++)
        {
            ang2vec(cp.thetas.p[k], 0, chord_pointing+3*k);
            ang2vec(cp.thetas.p[k] + PI/2, 0, dir1_proj_vec+3*k);
            cross(dir1_proj_vec+3*k, chord_pointing+3*k, dir2_proj_vec+3*k);
        }
        //accounting for CHORD's baseline shrinking when it points away from zenith
        float * L1s = new float [cp.thetas.l];
        for (unsigned int k = 0; k < cp.thetas.l; k++)
        {
            L1s[k] = cp.L1*cos(PI/180*(90-cp.CHORD_zenith_dec) - cp.thetas.p[k]);
        }

        float * threadu = u.p + (blockIdx.x*32 + threadIdx.x)*3;
	if (blockIdx.x*32 + threadIdx.x == 862) printf("u at pixel 45886: (%f,%f,%f) from device ID %d\n", threadu[0], threadu[1],threadu[2], deviceID);
	//if (blockIdx.x*32 + threadIdx.x == 862) printf("chord_pointing at pixel 45886: (%f,%f,%f)\n", chord_pointing[0], chord_pointing[1], chord_pointing[2]);
	for (unsigned int l = 0; l < wavelengths.l; l++)
        {
            float usum = 0;
            for (unsigned int s = 0; s*wavelengths.l < source_spectra.l; s++)
            {
		float time_sum = 0;
                if (source_spectra.p[s*wavelengths.l + l] > brightness_threshold)
                {
     		    float source_phi = atan2(source_positions.p[s*3+1],source_positions.p[s*3]);
		    float initial_travelangle = source_phi-cp.initial_phi_offset; //we want it to start computing phi_offset away from the source
                    if (deviceID == 1 && blockIdx.x*32 + threadIdx.x == 886) printf("initial travelangle: %f\n", initial_travelangle /PI*180);
		    for (unsigned int k = 0; k < cp.thetas.l; k++)
                    {
                        for (unsigned int j = 0; j < cp.time_samples; j++)
                        {
                            float travelangle = initial_travelangle+j*cp.delta_tau*omega;
			    if (deviceID == 1 && blockIdx.x*32 + threadIdx.x == 886) printf("offset phi: %f\n", travelangle/PI*180);
                            float u_rot [3];
                            rotate(threadu, u_rot, travelangle);
                            float source_rot [3];
                            rotate(source_positions.p+3*s, source_rot, travelangle);

                            float cdir1 = PI*L1s[k]/wavelengths.p[l]*subtractdot(source_rot, u_rot, dir1_proj_vec+3*k);
                            float cdir2 = PI*cp.L2 /wavelengths.p[l]*subtractdot(source_rot, u_rot, dir2_proj_vec+3*k);

			    //if (deviceID == 1 && blockIdx.x*32 + threadIdx.x == 886) printf("u_rot at pixel 45886: (%f,%f,%f)\n", u_rot[0], u_rot[1],u_rot[2]);
                            float Bsq_source = Bsq_from_vecs(source_rot, chord_pointing+3*k, wavelengths.p[l], cp.D);
                            float Bsq_u = Bsq_from_vecs(u_rot, chord_pointing+3*k, wavelengths.p[l], cp.D);

                            time_sum += Bsq_source * Bsq_u * sin_sq_ratio(cp.m1,cdir1) * sin_sq_ratio(cp.m2,cdir2);
			    //if (deviceID == 1 && blockIdx.x*32 + threadIdx.x == 886) printf("bsq_source, bsqu, and sinsqu parts: %e %e %e %e\n", Bsq_source, Bsq_u, sin_sq_ratio(cp.m1,cdir1), sin_sq_ratio(cp.m2,cdir2));
			    //if (deviceID == 1 && blockIdx.x*32 + threadIdx.x == 886) printf("Time sum at pixel 45886: %e\n", time_sum);
                        }
                    }
                }
                usum += source_spectra.p[s*wavelengths.l + l] * time_sum;
            }
            dm[(blockIdx.x*32 + threadIdx.x)*wavelengths.l + l] = usum;
            //if (deviceID == 1 && blockIdx.x*32 + threadIdx.x == 862) printf("Total sum at pixel 45886: %e from device ID %d\n", usum, deviceID);
        }
    delete chord_pointing;
    delete dir1_proj_vec;
    delete dir2_proj_vec;
    delete L1s;
    }
}

inline void copyFloatArrayToDevice (const floatArray host_array, floatArray & device_array)
{
    device_array.l = host_array.l;

    hipError_t err = hipSuccess;
    err = hipMalloc(&device_array.p, sizeof(float) * host_array.l);
    if (err != hipSuccess) {fprintf(stderr, "Failed to allocate memory for array (error code %s)!\n", hipGetErrorString(err)); exit(EXIT_FAILURE);}

    err = hipMemcpyAsync(device_array.p, host_array.p, sizeof(float) * host_array.l, hipMemcpyHostToDevice);
    if (err != hipSuccess) {fprintf(stderr, "Failed to copy data to device array (error code %s)!\n", hipGetErrorString(err)); exit(EXIT_FAILURE);}
}

extern "C" {void dirtymap_caller(const floatArray u, const floatArray wavelengths, const floatArray source_positions, const floatArray source_spectra, float brightness_threshold, const chordParams cp, float * dm)
{
    printf("initial_phi_offset cpu: %f\n", cp.initial_phi_offset);

    int deviceCount;
    hipGetDeviceCount(&deviceCount);
    std::cout << "Device count: " << deviceCount << std::endl;
    unsigned int npixels = u.l/3;
    //there are 4 GPUs, and each of them cover a quarter of the pixels
    unsigned int blocksToCover = (npixels+31)/32;
    unsigned int blocksPerGPU = (blocksToCover+deviceCount-1)/deviceCount;

    float * d_dm [deviceCount]; //array that holds pointers to the deviceCount output arrays
    floatArray d_u[deviceCount];
    floatArray d_wavelengths[deviceCount];
    floatArray d_source_positions[deviceCount];
    floatArray d_source_spectra[deviceCount];
    floatArray d_thetas[deviceCount];

    for (int gpuId = 0; gpuId < deviceCount; gpuId++)
    {
	hipSetDevice(gpuId);
	//copying data over to the device
        unsigned int npixels_per_gpu = ((gpuId+1) * blocksPerGPU * 32 <= npixels) ? blocksPerGPU * 32 : npixels - (deviceCount-1) * blocksPerGPU * 32;
	std::cout << "npixels_per_gpu for gpu " << gpuId << ": " << npixels_per_gpu << std::endl;
	floatArray u_for_gpu;
	u_for_gpu.p = u.p + (gpuId * blocksPerGPU * 32)*3;
	u_for_gpu.l = npixels_per_gpu*3;
	printf("The u 862 away from the pointer at %d (%f,%f,%f) (GPU %d)\n",(gpuId * blocksPerGPU * 32)*3, u_for_gpu.p[862*3],u_for_gpu.p[862*3+1],u_for_gpu.p[862*3+2],gpuId);
	copyFloatArrayToDevice(u_for_gpu,d_u[gpuId]);
	copyFloatArrayToDevice(wavelengths,d_wavelengths[gpuId]);
	copyFloatArrayToDevice(source_positions, d_source_positions[gpuId]);
	copyFloatArrayToDevice(source_spectra,d_source_spectra[gpuId]);
	copyFloatArrayToDevice(cp.thetas,d_thetas[gpuId]);

        //allocating the return array
        hipMalloc(&(d_dm[gpuId]), sizeof(float)*npixels_per_gpu*wavelengths.l);
    }

    //launching the kernels on all the GPUs
    for (int gpuId = 0; gpuId < deviceCount; gpuId++)
    {
	hipError_t cudaSetDeviceError;
	cudaSetDeviceError = hipSetDevice(gpuId);
	std::cout << "Error returned from cudasetdevice: " << cudaSetDeviceError << std::endl;
	int deviceId;
	hipGetDevice(&deviceId);

	chordParams d_cp = cp;
	printf("d_cp ipo: %f\n", d_cp.initial_phi_offset);
	d_cp.thetas = d_thetas[gpuId];

	dirtymap_kernel<<<blocksPerGPU,32>>>(d_u[gpuId], d_wavelengths[gpuId], d_source_positions[gpuId], d_source_spectra[gpuId], brightness_threshold, d_cp, d_dm[gpuId]);
	//cudaDeviceSynchronize();

	std::cout << "Ending loop for deviceId: " << deviceId << std::endl;
    }

    hipDeviceSynchronize();

    hipError_t kernel_err = hipGetLastError();
    if (kernel_err != hipSuccess) std::cout << "Error from kernel: " << kernel_err << std::endl;

    //copying over the data from the GPUs when they're done running
    for (int gpuId = 0; gpuId < deviceCount; gpuId++)
    {
	hipSetDevice(gpuId);
        unsigned int npixels_per_gpu = ((gpuId+1) * blocksPerGPU * 32 <= npixels) ? blocksPerGPU * 32 : npixels - (deviceCount-1) * blocksPerGPU * 32;
        hipMemcpyAsync(dm + gpuId * blocksPerGPU * 32 * wavelengths.l, d_dm[gpuId], sizeof(float)*npixels_per_gpu*wavelengths.l, hipMemcpyDeviceToHost);
        hipFree(d_dm[gpuId]);
	hipFree(d_u[gpuId].p);
	hipFree(d_wavelengths[gpuId].p);
	hipFree(d_source_positions[gpuId].p);
	hipFree(d_source_spectra[gpuId].p);
	hipFree(d_thetas[gpuId].p);
    }
    std::cout <<"(cpu) dirtymap at 45886: " << dm[45886] << std::endl;
}
}

int main ()
{
	size_t free, total;
	hipFree(0);
	hipMemGetInfo(&free,&total);
	std::cout << "free: " << free << " total: " << total << std::endl;
}
